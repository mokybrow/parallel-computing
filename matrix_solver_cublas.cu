#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>

#define N 3 // Размер вектора и матрицы
#define EPS 1e-3 // Точность
#define TAU 0.01 // Параметр метода

int main() {
    // Инициализация cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Объявление и инициализация массивов
    float h_A[N * N]; // матрица A
    float h_b[N]; // вектор b
    float h_xn[N] = {0.0f}; // начальное приближение (вектор из нулей)
    float h_xn1[N]; // следующее приближение
    float alpha = -TAU; // Коэффициент для выполнения ax - b
    float beta = 1.0f;  // для обновления

    // Инициализация матрицы A как единичной и вектора b
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * N + j] = (i == j) ? 1.0f : 0.0f; // оригинальная единичная матрица
        }
        h_b[i] = 1.0f; // Вектор b, состоящий из единиц
    }

    float *d_A, *d_b, *d_xn, *d_xn1;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_xn, N * sizeof(float));
    hipMalloc((void**)&d_xn1, N * sizeof(float));

    // Копирование данных на устройство
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_xn, h_xn, N * sizeof(float), hipMemcpyHostToDevice);

    float eps = 1e6; // Начальная невязка
    int num_iter = 0;

    while (eps > EPS) { // Цикл до достижения нужной точности
        // Выполнение матрично-векторного умножения: Ax (cuBLAS)
        hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, d_A, N, d_xn, 1, &beta, d_xn1, 1);
        
        // Добавляем вектор b к результату
        hipblasSaxpy(handle, N, &beta, d_b, 1, d_xn1, 1); // x_new = x_new + b (где alpha = 1.0)

        // Копирование следующего приближения с устройства на хост
        hipMemcpy(h_xn1, d_xn1, N * sizeof(float), hipMemcpyDeviceToHost);

        // Вычисление невязки
        eps = 0.0f;
        for (int i = 0; i < N; i++) {
            eps = fmaxf(eps, fabsf(h_xn1[i] - h_xn[i])); // Максимальный модуль разности
        }

        // Обновление текущего приближения на хосте
        for (int i = 0; i < N; i++) {
            h_xn[i] = h_xn1[i]; // Используйте новое приближение для следующей итерации
        }

        // Копирование нового приближения с хоста на устройство
        hipMemcpy(d_xn, h_xn, N * sizeof(float), hipMemcpyHostToDevice);
        num_iter++;

        printf("Iter: %d, EPS: %f\n", num_iter, eps);
    }

    // Копирование финального результата на хост
    hipMemcpy(h_xn, d_xn, N * sizeof(float), hipMemcpyDeviceToHost);

    // Печать результата
    printf("Final solution:\n");
    for (int i = 0; i < N; i++) {
        printf("x[%d] = %f\n", i, h_xn[i]);
    }

    // Освобождение памяти
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_xn);
    hipFree(d_xn1);
    hipblasDestroy(handle); // Освобождение ресурсов cuBLAS

    return 0;
}