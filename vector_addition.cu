#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10090000 // Размер векторов

// CUDA Kernel для сложения двух векторов
__global__ void addVectors(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Индекс текущего потока
    if (i < n) {
        C[i] = A[i] + B[i]; // Сложение векторов
    }
}

int main() {
    // Выделение памяти для векторов на хосте
    float *h_A = (float*)malloc(N * sizeof(float));
    float *h_B = (float*)malloc(N * sizeof(float));
    float *h_C = (float*)malloc(N * sizeof(float));

    // Инициализация векторов
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f; // Пример заполнения первого вектора
        h_B[i] = i * 2.0f; // Пример заполнения второго вектора
    }


    // Выделение памяти на устройстве
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // Копирование данных из хоста на устройство
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // Определение размеров блоков и сетки
    int threadsPerBlock = 256; // Количество потоков на блок
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; // Количество блоков

    // Вызов ядра
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Копирование результата с устройства на хост
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Вывод результата (например, первых 10 элементов)
    printf("Result of A + B:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);
    }

    // Освобождение памяти
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}