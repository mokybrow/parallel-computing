#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 3 // Размер вектора и матрицы
#define EPS 1e-3 // Точность
#define TAU 0.01 // Параметр метода

// CUDA Kernel для вычисления следующего приближения
__global__ void computeNextApproximation(float *A, float *b, float *xn, float *xn1) {
    int index = threadIdx.x;
    if (index < N) {
        float Ax = 0.0;
        for (int j = 0; j < N; j++) {
            Ax += A[index * N + j] * xn[j]; // Вычисление A*x
        }
        xn1[index] = xn[index] - TAU * (Ax - b[index]); // Обновление следующего приближения
    }
}

int main() {
    // Объявление и инициализация массивов
    float h_A[N * N]; // матрица A
    float h_b[N]; // вектор b
    float h_xn[N] = {0.0f}; // начальное приближение (вектор из нулей)
    float h_xn1[N]; // следующее приближение

    // Инициализация матрицы A как единичной и вектора b
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * N + j] = (i == j) ? 1.0f : 0.0f; // оригинальная единичная матрица
        }
        h_b[i] = 1.0f; // Вектор b, состоящий из единиц
    }

    float *d_A, *d_b, *d_xn, *d_xn1;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_xn, N * sizeof(float));
    hipMalloc((void**)&d_xn1, N * sizeof(float));

    // Копирование данных на устройство
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_xn, h_xn, N * sizeof(float), hipMemcpyHostToDevice);

    float eps = 1e6; // Начальная невязка
    int num_iter = 0;

    while (eps > EPS) { // Цикл до достижения нужной точности
        // Вызов ядра
        computeNextApproximation<<<1, N>>>(d_A, d_b, d_xn, d_xn1);
        hipDeviceSynchronize();

        // Копирование следующего приближения с устройства на хост
        hipMemcpy(h_xn1, d_xn1, N * sizeof(float), hipMemcpyDeviceToHost);

        // Вычисление невязки
        eps = 0.0f;
        for (int i = 0; i < N; i++) {
            eps = fmaxf(eps, fabsf(h_xn1[i] - h_xn[i])); // Максимальный модуль разности
        }

        // Обновление текущего приближения на хосте
        for (int i = 0; i < N; i++) {
            h_xn[i] = h_xn1[i]; // Используйте новое приближение для следующей итерации
        }

        // Копирование нового приближения с хоста на устройство
        hipMemcpy(d_xn, h_xn, N * sizeof(float), hipMemcpyHostToDevice);
        num_iter++;

        printf("Iter: %d, EPS: %f\n", num_iter, eps);
    }

    // Копирование финального результата на хост
    hipMemcpy(h_xn, d_xn, N * sizeof(float), hipMemcpyDeviceToHost);

    // Печать результата
    printf("Final solution:\n");
    for (int i = 0; i < N; i++) {
        printf("x[%d] = %f\n", i, h_xn[i]);
    }

    // Освобождение памяти
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_xn);
    hipFree(d_xn1);

    return 0;
}