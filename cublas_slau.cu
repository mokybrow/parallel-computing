
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

void initializeMatrix(float* A, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        A[i] = (float)(rand() % 10); // инициализация случайными числами от 0 до 9
    }
}

void printMatrix(float* C, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", C[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int M = 2; // количество строк в матрице A
    int N = 3; // количество столбцов в матрице B
    int K = 4; // количество столбцов в матрице A и строк в матрице B

    // Выделяем память для матриц на хосте
    float *h_A = (float*)malloc(M * K * sizeof(float));
    float *h_B = (float*)malloc(K * N * sizeof(float));
    float *h_C = (float*)malloc(M * N * sizeof(float));

    // Инициализируем матрицы A и B
    initializeMatrix(h_A, M, K);
    initializeMatrix(h_B, K, N);

    // Вывод матриц
    printf("Matrix A:\n");
    printMatrix(h_A, M, K);
    printf("Matrix B:\n");
    printMatrix(h_B, K, N);

    // Выделяем память для матриц на устройстве
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // Копируем матрицы A и B на устройство
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Создаем дескриптор cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Умножаем матрицы A и B, результат сохраняется в C
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                M, N, K, 
                &alpha, 
                d_A, M,   // матрица A
                d_B, K,   // матрица B
                &beta, 
                d_C, M);  // результат C

    // Копируем результат обратно на хост
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Вывод результата
    printf("Matrix C (result of A * B):\n");
    printMatrix(h_C, M, N);

    // Освобождаем память
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}